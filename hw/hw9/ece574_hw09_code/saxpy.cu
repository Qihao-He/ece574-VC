/* Based on code from here: http://devblogs.nvidia.com/parallelforall/easy-introduction-cuda-c-and-c/ */



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (1000*1000*8)

/* Calculate SAXPY, single-precision vector math 	*/
/* y[i]=a*x[i]+y[i]					*/

__global__
void saxpy (int n, float a, float *x, float *y) {

	int i=blockIdx.x*blockDim.x+threadIdx.x;

	/* Only run calculation if we are in range */
	/* where i is valid.  It can be out of range */
	/* if our vector is shorter than a */
	/* multiple of the blocksize */

	if (i<n) {
		y[i]=a*x[i]+y[i];
	}
}

int main(int argc, char **argv) {

	int i;
	float *x, *y, *dev_x, *dev_y;
	float a;

	float loops=1.0;

	if (argc>1) {
		loops=atof(argv[1]);
	}

	/* Allocate vectors on CPU */
	x=(float *)malloc(N*sizeof(float));
	y=(float *)malloc(N*sizeof(float));

	/* Allocate vectors on GPU */
	hipMalloc((void **)&dev_x,N*sizeof(float));
	hipMalloc((void **)&dev_y,N*sizeof(float));

	/* Initialize the host vectors */
	for(i=0;i<N;i++) {
		x[i]=(float)i;
		y[i]=(float)(10.0*i);
	}

	hipMemcpy(dev_x,x,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,N*sizeof(float),hipMemcpyHostToDevice);

	printf("Size: %d\n",(N+255)/256);

	/* Perform SAXPY */
	for(a=0;a<loops;a+=1.0) {
		saxpy<<<(N+255)/256,256>>>(N,a,dev_x,dev_y);
	}

	hipMemcpy(y,dev_y,N*sizeof(float),hipMemcpyDeviceToHost);

	/* results */
	i=100;
	printf("y[%d]=%f %f\n",i,y[i],y[i+1]);

	/* 0: a=0, x=100, y=1000 ::::::: y=1000 */
	/* 1: a=1, x=100, y=1000 ::::::: y=1100 */
	/* 2: a=2, x=100, y=1100 ::::::: y=1300 */
	/* 3: a=3, x=100, y=1300 ::::::: y=1600 */
	/* 4: a=4, x=100, y=1600 ::::::: y=2000 */

	hipFree(dev_x);
	hipFree(dev_y);

	return 0;
}

