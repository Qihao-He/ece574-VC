#include "hip/hip_runtime.h"
/* Example sobel code for ECE574 -- Spring 2017 */
/* By Vince Weaver <vincent.weaver@maine.edu> */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <errno.h>
#include <math.h>

#include <jpeglib.h>

#include <hip/hip_runtime.h>

#include <papi.h>

/* Filters */
static int sobel_x_filter[3][3]={{-1,0,+1},{-2,0,+2},{-1,0,+1}};
static int sobel_y_filter[3][3]={{-1,-2,-1},{0,0,0},{1,2,+1}};

/* Structure describing the image */
struct image_t {
	int x;
	int y;
	int depth;	/* bytes */
	unsigned char *pixels;
};

struct convolve_data_t {
	struct image_t *old;
	struct image_t *newt;
	int (*filter)[3][3];
	int ystart;
	int yend;
};

/* For the generic convolve, you will also need to upload the sobelx and sobely
matrices to the device. A simple array of 9 ints is probably best. */
__global__ //fine grained
void cuda_generic_convolve (int n,int x,int y,unsigned char *in,int *matrix,unsigned char *out) {
	//Can get block number with blockIdx.x and thread index with threadIdx.x
	/* The hardest part here is getting the grid/block/thread count right.s */
	int blockId=blockIdx.y * gridDim.x + blockIdx.x;
	int i=blockId * blockDim.x + threadIdx.x;//thread index

/* Remember there are separate RGB colors so you will need to add in points -3,
0, +3 for example */
/* Also make sure you have code that skips the first and last rows, as well as
the first and last columns (which is three columns, remember RGB). */
	// if(i<x*3 || i>=(y-1)*x*3){}//filter out 1st last rows, 1st 3 and last 3 columns
	// else if(i%(3*x)<3 || i%(3*x)>((3*x)-3)){}
	// else{/* For each point “i” add in the 9 scaled values. */
	// 	out[i]=in[i]*matrix[0];
	// 	out[i]+=in[i]*matrix[1];
	// 	out[i]+=in[i]*matrix[2];
	// 	out[i]+=in[i]*matrix[3];
	// 	out[i]+=in[i]*matrix[4];
	// 	out[i]+=in[i]*matrix[5];
	// 	out[i]+=in[i]*matrix[6];
	// 	out[i]+=in[i]*matrix[7];
	// 	out[i]+=in[i]*matrix[8];
	// }
	out[i]=0xff;//test on all white

/* Again it might be helpful to output the sobel_x output and run on the
butterfinger input and getthat to match exactly before running with both sobel_y
 and combine hooked up. */
}

//some noise pixels
__global__ //coarse grained
void cuda_combine (int n, unsigned char *in_x,unsigned char *in_y,unsigned char *out) {

	int i=blockIdx.x*blockDim.x+threadIdx.x;//thread index
	out[i]=sqrt(double(
		(in_x[i]*in_x[i])+
		(in_y[i]*in_y[i])
	));
	if (out[i]>255) out[i]=255;
	// if (out[i]<0) out[i]=0;//not necessary
	// out[i]=0xff;
}

/* very inefficient convolve code */
// static void *generic_convolve(void *argument) {
//
// 	int x,y,k,l,d;
// 	uint32_t color;
// 	int sum,depth,width;
//
// 	struct image_t *old;
// 	struct image_t *newt;
// 	int (*filter)[3][3];
// 	struct convolve_data_t *data;
// 	int ystart, yend;
//
// 	/* Convert from void pointer to the actual data type */
// 	data=(struct convolve_data_t *)argument;
// 	old=data->old;
// 	newt=data->newt;
// 	filter=data->filter;
//
// 	ystart=data->ystart;
// 	yend=data->yend;
//
// 	depth=old->depth;
// 	width=old->x*old->depth;
//
// 	if (ystart==0) ystart=1;
// 	if (yend==old->y) yend=old->y-1;
//
// 	for(d=0;d<3;d++) {
// 	   for(x=1;x<old->x-1;x++) {
// 	     for(y=ystart;y<yend;y++) {
// 		sum=0;
// 		for(k=-1;k<2;k++) {
// 		   for(l=-1;l<2;l++) {
// 			color=old->pixels[((y+l)*width)+(x*depth+d+k*depth)];
// 			sum+=color * (*filter)[k+1][l+1];
// 		   }
// 		}
//
// 		if (sum<0) sum=0;
// 		if (sum>255) sum=255;
//
// 		newt->pixels[(y*width)+x*depth+d]=sum;
// 	     }
// 	   }
// 	}
//
// 	return NULL;
// }

// static int combine(struct image_t *s_x,
// 			struct image_t *s_y,
// 			struct image_t *newt) {
// 	int i;
// 	int out;
//
// 	for(i=0;i<( s_x->depth * s_x->x * s_x->y );i++) {
//
// 		out=sqrt(
// 			(s_x->pixels[i]*s_x->pixels[i])+
// 			(s_y->pixels[i]*s_y->pixels[i])
// 			);
// 		if (out>255) out=255;
// 		if (out<0) out=0;
// 		newt->pixels[i]=out;
// 	}
//
// 	return 0;
// }

static int load_jpeg(char *filename, struct image_t *image) {

	FILE *fff;
	struct jpeg_decompress_struct cinfo;
	struct jpeg_error_mgr jerr;
	JSAMPROW output_data;
	unsigned int scanline_len;
	int scanline_count=0;

	fff=fopen(filename,"rb");
	if (fff==NULL) {
		fprintf(stderr, "Could not load %s: %s\n",
			filename, strerror(errno));
		return -1;
	}

	/* set up jpeg error routines */
	cinfo.err = jpeg_std_error(&jerr);

	/* Initialize cinfo */
	jpeg_create_decompress(&cinfo);

	/* Set input file */
	jpeg_stdio_src(&cinfo, fff);

	/* read header */
	jpeg_read_header(&cinfo, TRUE);

	/* Start decompressor */
	jpeg_start_decompress(&cinfo);

	printf("output_width=%d, output_height=%d, output_components=%d\n",
		cinfo.output_width,
		cinfo.output_height,
		cinfo.output_components);

	image->x=cinfo.output_width;
	image->y=cinfo.output_height;
	image->depth=cinfo.output_components;

	scanline_len = cinfo.output_width * cinfo.output_components;
	image->pixels=(unsigned char *)malloc(cinfo.output_width * cinfo.output_height * cinfo.output_components);

	while (scanline_count < cinfo.output_height) {
		output_data = (image->pixels + (scanline_count * scanline_len));
		jpeg_read_scanlines(&cinfo, &output_data, 1);
		scanline_count++;
	}

	/* Finish decompressing */
	jpeg_finish_decompress(&cinfo);

	jpeg_destroy_decompress(&cinfo);

	fclose(fff);

	return 0;
}

static int store_jpeg(const char *filename, struct image_t *image) {

	struct jpeg_compress_struct cinfo;
	struct jpeg_error_mgr jerr;
	int quality=90; /* % */
	int i;

	FILE *fff;

	JSAMPROW row_pointer[1];
	int row_stride;

	/* setup error handler */
	cinfo.err = jpeg_std_error(&jerr);

	/* initialize jpeg compression object */
	jpeg_create_compress(&cinfo);

	/* Open file */
	fff = fopen(filename, "wb");
	if (fff==NULL) {
		fprintf(stderr, "can't open %s: %s\n",
			filename,strerror(errno));
		return -1;
	}

	jpeg_stdio_dest(&cinfo, fff);

	/* Set compression parameters */
	cinfo.image_width = image->x;
	cinfo.image_height = image->y;
	cinfo.input_components = image->depth;
	cinfo.in_color_space = JCS_RGB;
	jpeg_set_defaults(&cinfo);
	jpeg_set_quality(&cinfo, quality, TRUE);

	/* start compressing */
	jpeg_start_compress(&cinfo, TRUE);

	row_stride=image->x*image->depth;

	for(i=0;i<image->y;i++) {
		row_pointer[0] = & image->pixels[i * row_stride];
		jpeg_write_scanlines(&cinfo, row_pointer, 1);
	}

	/* finish compressing */
	jpeg_finish_compress(&cinfo);

	/* close file */
	fclose(fff);

	/* clean up */
	jpeg_destroy_compress(&cinfo);

	return 0;
}

int main(int argc, char **argv) {

	struct image_t image,sobel_x,sobel_y,new_image;
	// struct convolve_data_t sobel_data[2];
	long long start_time,load_time,convolve_time;
	long long combine_after,combine_before;
	long long copy_before,copy_after,copy2_before,copy2_after;
	long long store_after,store_before;

	long long cudaMalloc_after,cudaMalloc_before,cudaMalloc2_after,cudaMalloc2_before;

	unsigned char *dev_x, *dev_y,*out;// Pointer to host & device arrays
	int *dev_x_filter, *dev_y_filter;// Pointer to host & device arrays
	long long n;// Number of pixels in a picture

	/* Check command line usage */
	if (argc<2) {
		fprintf(stderr,"Usage: %s image_file\n",argv[0]);
		return -1;
	}

	PAPI_library_init(PAPI_VER_CURRENT);

	start_time=PAPI_get_real_usec();

	/* Load an image */
	load_jpeg(argv[1],&image);

	load_time=PAPI_get_real_usec();

/* Allocate device buffers for sobelx, sobely, and the output using hipMalloc() */
	/* Allocate space for output image */
	new_image.x=image.x;
	new_image.y=image.y;
	new_image.depth=image.depth;
	new_image.pixels=(unsigned char *)malloc(image.x*image.y*image.depth*sizeof(char));
	// new_image.pixels=(unsigned char *)hipMalloc(image.x*image.y*image.depth*sizeof(char));

	/* Allocate space for output image */
	sobel_x.x=image.x;
	sobel_x.y=image.y;
	sobel_x.depth=image.depth;
	sobel_x.pixels=(unsigned char *)malloc(image.x*image.y*image.depth*sizeof(char));
	// sobel_x.pixels=(unsigned char *)hipMalloc(image.x*image.y*image.depth*sizeof(char));

	/* Allocate space for output image */
	sobel_y.x=image.x;
	sobel_y.y=image.y;
	sobel_y.depth=image.depth;
	sobel_y.pixels=(unsigned char *)malloc(image.x*image.y*image.depth*sizeof(char));
	// sobel_y.pixels=(unsigned char *)hipMalloc(image.x*image.y*image.depth*sizeof(char));

	n=image.x*image.y*image.depth*sizeof(char);//number of pixels of the picture

/* Allocate arrays on GPU */
	cudaMalloc_before=PAPI_get_real_usec();
	hipMalloc((void**)&dev_x,n*sizeof(unsigned char));
	hipMalloc((void**)&dev_y,n*sizeof(unsigned char));
	hipMalloc((void**)&dev_x_filter,9*sizeof(int));
	hipMalloc((void**)&dev_y_filter,9*sizeof(int));
	cudaMalloc_after=PAPI_get_real_usec();

/* Copy the local sobel_x.pixels and sobel_y.pixels to the device using hipMemcpy() */
	copy_before=PAPI_get_real_usec();
	hipMemcpy(dev_x,image.pixels,n*sizeof(unsigned char),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,image.pixels,n*sizeof(unsigned char),hipMemcpyHostToDevice);
	hipMemcpy(dev_x_filter,sobel_x_filter,9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_y_filter,sobel_y_filter,9*sizeof(int),hipMemcpyHostToDevice);
	copy_after=PAPI_get_real_usec();

/* PERFORM KERNEL: cuda_generic_convolve */
	/* convolution */
	// sobel_data[0].old=&image;
	// sobel_data[0].newt=&sobel_x;
	// sobel_data[0].filter=&sobel_x_filter;
	// sobel_data[0].ystart=0;
	// sobel_data[0].yend=image.y;
	// generic_convolve((void *)&sobel_data[0]);
	// cuda_generic_convolve (int n, char *in, int *matrix, char *out)
	// first inside brackets is number of blocks, second is threads per block
	cuda_generic_convolve<<<(n+256)/256, 256>>>(n,image.x,image.y,sobel_x.pixels,dev_x_filter,dev_x);

	// sobel_data[1].old=&image;
	// sobel_data[1].newt=&sobel_y;
	// sobel_data[1].filter=&sobel_y_filter;
	// sobel_data[1].ystart=0;
	// sobel_data[1].yend=image.y;
	// generic_convolve((void *)&sobel_data[1]);
	cuda_generic_convolve<<<(n+256)/256, 256>>>(n,image.x,image.y,sobel_y.pixels,dev_y_filter,dev_y);

	// make the host block until the device is finished
	hipDeviceSynchronize();
	convolve_time=PAPI_get_real_usec();

/* Allocate arrays on GPU */
	cudaMalloc2_before=PAPI_get_real_usec();
	hipMalloc((void**)&out,n*sizeof(unsigned char));
	cudaMalloc2_after=PAPI_get_real_usec();

	/*  Some hints: to debug that your kernel works, you can first set all output to 0xff and verify you get an all-white image back. */
	// new_image.pixels=0xff;

	/* Combine to form output */
	// combine(&sobel_x,&sobel_y,&new_image);
	// cuda_combine (int n, unsigned char *in_x,	unsigned char *in_y, unsigned char *out)
	// first inside brackets is number of blocks, second is threads per block
	combine_before=PAPI_get_real_usec();
	cuda_combine<<<(n+256)/256, 256>>>(n,dev_x,dev_y,out);
	combine_after=PAPI_get_real_usec();

	/* Copy the results back into new_image.pixels using hipMemcpy() (be sure to get the direction right) */
	copy2_before=PAPI_get_real_usec();
	hipMemcpy(new_image.pixels,out,n*sizeof(unsigned char),hipMemcpyDeviceToHost);
	copy2_after=PAPI_get_real_usec();

	/* REPLACE THE ABOVE WITH YOUR CODE */
	/* IT SHOULD ALLOCATE SPACE ON DEVICE */
	/* COPY SOBEL_X and SOBEL_Y data to device */
	/* RUN THE KERNEL */
	/* THEN COPY THE RESULTS BACK */

	/* Write data back out to disk */
	store_before=PAPI_get_real_usec();
	store_jpeg("out.jpg",&new_image);
	store_after=PAPI_get_real_usec();

	/* Print timing results */
	printf("Load time: %lld\n",load_time-start_time);
  printf("Convolve time: %lld\n",convolve_time-load_time);
	printf("hipMalloc time: %lld\n",cudaMalloc_after-cudaMalloc_before+cudaMalloc2_after-cudaMalloc2_before);
	printf("Copy time: %lld\n",(copy_after-copy_before)+(copy2_after-copy2_before));
  printf("Combine time: %lld\n",combine_after-combine_before);
  printf("Store time: %lld\n",store_after-store_before);
	printf("Total time = %lld\n",store_after-start_time);

	hipFree(dev_x);//hipFree device name
	hipFree(dev_y);
	hipFree(dev_x_filter);
	hipFree(dev_y_filter);
	hipFree(out);

	return 0;
}
